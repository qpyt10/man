" vi map for CUDA
"
" 12.10.11
" 11.12.12 m
"
"
" use ,,cu to load
" use ,[name]
"
"

map! -ret( <ESC>?(<CR>:silent noh<CR>:<CR><RIGHT>
map! -ret& <ESC>?&<CR>:silent noh<CR>:<CR><RIGHT>
map! -retSPACE <ESC>?\ <CR>:silent noh<CR>:<CR><RIGHT>
map! -ret0 <ESC>^

map! -dpnt d_pnt
map! -hpnt h_pnt
map! -dev dev
map! -src src
map! -dst dst
map! -num num
map! -dir dir
map! -str Stream
map! -evn Event
map! -pro prop
map! -time Etime
map! -start start
map! -end end

" index "
map! ,tix threadIdx.x
map! ,tiy threadIdx.y
map! ,bix blockIdx.x
map! ,biy blockIdx.y
map! ,bdx blockDim.x
map! ,bdy blockDim.y


" kernel launch "
map! ,ker kernel<<<numBlock, numThread, constMemory, stream>>>();-ret0

" device managment "
map! ,dcou hipGetDeviceCount(&-dev);-ret&
map! ,dset hipSetDevice(-dev);-ret(
map! ,dget hipGetDevice(&-dev);-ret&
map! ,dsyn hipDeviceSynchronize();<CR>
map! ,dpro hipGetDeviceProperties(&-pro, -dev)-ret&

" memory managment "
map! ,cmal  hipMalloc( (void**) &-dpnt, -num);-ret&
map! ,cmalh hipHostMalloc( (void**) &-hpnt, -num);-ret&
map! ,cfre  hipFree(-dpnt);-ret(
map! ,cfreh hipHostFree(-hpnt);-ret(

" memory copy "
map! ,cp   hipMemcpy(-dst, -src, -num, -dir);-ret
map! ,acp  hipMemcpyAsync(-dst, -src, -num, -dir, -str);-ret

map! ,h2h  hipMemcpyHostToHost
map! ,h2d  hipMemcpyHostToDevice
map! ,d2h  hipMemcpyDeviceToHost
map! ,d2d  hipMemcpyDeviceToDevice

" strream managment "
map! ,str  hipStream_t ;-retSPACE
map! ,scre hipStreamCreate(&-str);-ret&
map! ,sdes hipStreamDestroy(-str);-ret(
map! ,ssyn hipStreamSynchronize(-str);-ret(

" event managment "
map! ,ev   hipEvent_t ;-retSPACE
map! ,ecre hipEventCreate(&-evn);-ret&
map! ,edes hipEventDestroy(-evn);-ret(
map! ,esyn hipEventSynchronize(-evn);-ret(
map! ,erec hipEventRecord(-evn, -str);-ret(
map! ,etim hipEventElapsedTime(&-time, -start, -end);-ret&

" peer2peer "
map! ,pcan hipDeviceCanAccessPeer(&can, from, to);-ret&
map! ,pena hipDeviceEnablePeerAccess(-dev, 0);-ret(
map! ,pdis hipDeviceDisablePeerAccess(-dev);-ret(
map! ,pcp  hipMemcpyPeer(-dst, -dst-dev, -src, -src-dev, -num);-ret(
map! ,pcpa hipMemcpyPeerAsync(-dst, -dst-dev, -src, -src-dev, -num, -str);-ret(

:function! Clean()
: unmap! ,str
:endfunction
